#include "hip/hip_runtime.h"
// CUDA Dot Product Program
// CS 3220
// Authors: Kai Pasciak, Walter Clay

#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

#define N 65536 // 256 * 256

__global__
void dotp( float *u, float *v, float *partialSums, int n ){
    __shared__ float localCache[BLOCK_SIZE];

    // Compute localCache[i]
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    localCache[threadIdx.x = U[tidx] * V[tidx];

    // Synchronize Threads
    __syncthreads();

    // Sum localCache using parallel reduction
    cacheIndex = threadIdx.x;
    int i = blockDim.x / 2;
    while (i > 0){
        if (cacheIndex < i)
            localCache[cacheindex] = localCache[cacheIndex] + localCache[cacheIndex + i];
        __syncthreads();
        i = i / 2;
    }

    if (cacheIndex == 0)
        partialSum[blockIdx.x] = localCache[cacheIdx];
}

int main(){
    // Initialize and define constants
    int numBlocks = 256;
    int threadsPerBlock = 256;

    // Initialize variables
    float *U, *V, *partialSum;
    float dev_U, dev_V;

    // Allocate memory on the host
    U = (float *) malloc(N * sizeof(float));
    V = (float *) malloc(N * sizeof(float));
    partialSum = (float *) malloc(numBlocks * sizeof(float));

    // Allocate memory on the device
    hipMalloc( (void **) &dev_U, N*sizeof(float) );
    hipMalloc( (void **) &dev_V, N*sizeof(float) );


    // Set up problem on the host
    // Create seed for random number generator
    srand48(time(nullptr));

    // Set vector contents to random numbers
    for (int i = 0; i < N; ++i) {
        float randomU == drand48();
        U[i] = random U;

        float randomV == drand48();
        V[i] = random V;
    }

    // Start timer for including memory copies
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    // Copy data to the GPU
    hipMemcpy( dev_U, U, N*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_V, V, N*sizeof(float), hipMemcpyHostToDevice );

    // Start timer not including memory copies
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2,0);

    // Call kernel
    dotp<<<numBlocks, threadsPerBlock>>>( dev_U, dev_V, dev_partialSum, N );

    // Synchronize
    hipDeviceSynchronize();

    // End timer not including memory copies
    hipEventRecord(stop2, 0);

    // Copy results to host
    hipMemcpy( partialSum, dev_partialSum, numBlocks*sizeof(float), cudaMemCpyDeviceToHost);

    // End timer including memory copies
    hipEventRecord(stop, 0);

    // Calculate elapsed time including memory copies
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Calculate elapsed time  not including memory copies
    float elapsedTime2;
    hipEventElapsedTime(&elapsedTime2, start2, stop2);

    // Sum partial sums
    float gpuResult = 0.0;
    for (int i = 0; i < numBlocks; ++i)
        gpuResult = gpuResult + partialSum[i];

    // TODO: Implement CPU calculation and time

    // TODO: Calculate relative error

    // Clean up
    hipFree( dev_U );
    hipFree( dev_V );

    free(U);
    free(V);
    free(partialSum);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}