// CUDA Matrix Vector Product Program
// CS 3220
// Authors: Kai Pasciak, Walter Clay

#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <ctime>
#include <iostream>
#include <hip/hip_runtime.h>


#define N 5000  // Dimension of the matrix and vector
#define THREADS_PER_BLOCK 256  // Number of threads per block

using namespace std;

// CUDA kernel for performing matrix-vector multiplication
__global__ void MxV(float *M, float *x, float *y, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (row < n) {
        float sum = 0.0;
        for (int j = 0; j < n; j++) {
            sum += M[row * n + j] * x[j];  // Perform the dot product
        }
        y[row] = sum;  // Store the result in y
    }
}

// CPU function for matrix-vector multiplication
void matrixVectorProductCPU(float *M, float *x, float *y, int n) {
    for (int i = 0; i < n; ++i) {
        y[i] = 0.0;
        for (int j = 0; j < n; ++j) {
            y[i] += M[i * n + j] * x[j];
        }
    }
}

// Function to calculate the Euclidean norm of a vector
float vectorNorm(float *v, int n) {
    float sum = 0.0;
    for (int i = 0; i < n; ++i) {
        sum += v[i] * v[i];
    }
    return sqrt(sum);
}

// Function to compute the absolute difference vector between two vectors
void vectorDifference(float *v1, float *v2, float *diff, int n) {
    for (int i = 0; i < n; ++i) {
        diff[i] = abs(v1[i] - v2[i]);
    }
}

int main() {
    float *M, *x, *y, *y_cpu, *diff;
    float *dev_M, *dev_x, *dev_y;
    int size = N * N * sizeof(float);
    int vectorSize = N * sizeof(float);

    // Allocate memory on the host
    M = (float *)malloc(size);
    x = (float *)malloc(vectorSize);
    y = (float *)malloc(vectorSize);
    y_cpu = (float *)malloc(vectorSize);
    diff = (float *)malloc(vectorSize);

    // Allocate memory on the device
    hipMalloc(&dev_M, size);
    hipMalloc(&dev_x, vectorSize);
    hipMalloc(&dev_y, vectorSize);

    // Initialize matrix M and vector x with random values
    srand(static_cast<unsigned int>(time(0)));
    for (int i = 0; i < N * N; i++) {
        M[i] = rand() / (float)RAND_MAX;
    }
    for (int i = 0; i < N; i++) {
        x[i] = rand() / (float)RAND_MAX;
    }

    // Copy data from host to device
    hipMemcpy(dev_M, M, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, vectorSize, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blocks((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threads(THREADS_PER_BLOCK);

    // Setup timing variables for GPU computation excluding memory copy
    hipEvent_t startCompute, stopCompute;
    hipEventCreate(&startCompute);
    hipEventCreate(&stopCompute);
    hipEventRecord(startCompute);

    // Launch the kernel
    MxV<<<blocks, threads>>>(dev_M, dev_x, dev_y, N);
    hipDeviceSynchronize();  // Ensure all threads have finished

    // Stop timing after synchronization
    hipEventRecord(stopCompute);
    hipEventSynchronize(stopCompute);

    // Measure elapsed time for the computation only
    float computeMilliseconds = 0;
    hipEventElapsedTime(&computeMilliseconds, startCompute, stopCompute);
    cout << "GPU Compute Time (excluding memory copies): " << computeMilliseconds << " ms" << endl;

    // Copy result back to host
    hipMemcpy(y, dev_y, vectorSize, hipMemcpyDeviceToHost);

    // Perform computation on the CPU
    matrixVectorProductCPU(M, x, y_cpu, N);

    // Compute the difference vector and norms for relative error calculation
    vectorDifference(y, y_cpu, diff, N);
    float diffNorm = vectorNorm(diff, N);
    float yCpuNorm = vectorNorm(y_cpu, N);

    // Compute relative error
    float relativeError = diffNorm / yCpuNorm;
    cout << "Relative Error: " << relativeError << endl;

    // Clean up
    hipFree(dev_M);
    hipFree(dev_x);
    hipFree(dev_y);
    free(M);
    free(x);
    free(y);
    free(y_cpu);
    free(diff);

    hipEventDestroy(startCompute);
    hipEventDestroy(stopCompute);

    return 0;
}

